// Author: Noah Van Der Weide
// main.cu for image-upscaling with CUDA
//
// 1. take in user specified PPM image
// 2. determine width and height
// 3. convert to vector/array
// 4. create new vector/array with larger dimensions
// 5. call upscale.cu function
// 6. export new upscaled image and time to complete
// 7. clean up (if needed)

#include "upscale.cuh"
#include <iostream>


int main (int argc, char * argv[]){

  // CUDA timing parameters
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float ms;




  hipEventRecord(start);
  // upscale.cu here
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);

  std::cout << "\ntime (ms) = " << ms << std::endl;
  // clean up (if needed)

  return 0;
}
