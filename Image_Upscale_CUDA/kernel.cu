#include "hip/hip_runtime.h"
// Author: Noah Van Der Weide
#include "hip/hip_runtime.h"
#include ""
#include "Upscale_CUDA.h"
#include <stdlib.h>
#include <math.h>
#include <iostream>
//#define THREADS_PER_BLOCK 64

__global__ void stretch_CUDA(unsigned char* dst, unsigned char* src, int src_width, int src_height, int channels, unsigned int threshold);
__global__ void fill_CUDA(unsigned char* dst, int dst_width, int src_height, int channels, unsigned int threshold);

void upscale(unsigned char* src, unsigned char* dst, int src_height, int src_width, int dst_height, int dst_width, int channels, unsigned int threshold) {

	// initialize device variables
	unsigned char* dev_src, * dev_dst;

	// CUDA timing parameters
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float ms;

	// number of elements (if a picture has 3 channels, this is 3 * pixels)
	int dst_elements = dst_width * dst_height * channels;
	int src_elements = src_width * src_height * channels;

	// allocate memory in GPU
	hipMalloc((void**)&dev_dst, dst_elements);
	hipMalloc((void**)&dev_src, src_elements);

	// copy data from CPU to GPU
	hipMemcpy(dev_dst, dst, dst_elements, hipMemcpyHostToDevice);
	hipMemcpy(dev_src, src, src_elements, hipMemcpyHostToDevice);

	// start timer for performance evaluation
	hipEventRecord(start);

	// call upscale function
	dim3 src_grid((src_width + 31) / 32, (src_height + 31) / 32);
	dim3 dst_grid((dst_width + 31) / 32, (src_height + 31) / 32);
	dim3 blocks(32, 32);

	stretch_CUDA << < src_grid, blocks >> > (dev_dst, dev_src, src_width, src_height, channels, threshold);
	hipDeviceSynchronize();
	fill_CUDA << < dst_grid, blocks >> > (dev_dst, dst_width, src_height, channels, threshold);
	hipDeviceSynchronize();

	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);

	// copy data back from GPU to CPU
	hipMemcpy(dst, dev_dst, dst_elements, hipMemcpyDeviceToHost);

	// display time
	std::cout << "Upscale function finished in " << ms << " ms" << std::endl;

	// free GPU
	hipFree(dev_dst);
	hipFree(dev_src);
}

__global__ void stretch_CUDA(unsigned char* dst, unsigned char* src, int src_width, int src_height, int channels, unsigned int threshold) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= src_width || y >= src_height)
		return;

	int dst_width = src_width * 3 - 2;
	int dst_index = ((x * 3) + (y * dst_width * 3)) * channels;
	int src_index = (x + y * src_width) * channels;

	int k;

	unsigned int diff = 0;
	unsigned int temp = 0;

	// check every channel for differences. If just one of the channels has a difference above the threshold, then apply nearest neighbor. 

	// horizontal
	for (k = 0; k < channels; k++) {
		temp = abs(src[src_index + channels + k] - src[src_index + k]); // difference between two color channels
		if (temp > diff)
			diff = temp;
	}

	if (diff > threshold) { // nearest neighbor
		for (k = 0; k < channels; k++) {
			dst[dst_index + k] = src[src_index + k];
			dst[dst_index + channels + k] = src[src_index + k];
			dst[dst_index + 2 * channels + k] = src[src_index + channels + k];
		}
	}
	else { // linear
		int step;
		for (k = 0; k < channels; k++) {
			step = (src[src_index + k] - src[src_index + channels + k]) / 3;
			dst[dst_index + k] = src[src_index + k];
			dst[dst_index + channels + k] = src[src_index + k] - step;
			dst[dst_index + 2 * channels + k] = src[src_index + k] - (2 * step);
		}
	}
}

__global__ void fill_CUDA(unsigned char* dst, int dst_width, int src_height, int channels, unsigned int threshold) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= dst_width || y >= src_height)
		return;

	int dst_index = (x + (y * dst_width * 3)) * channels;

	int dst_stride = dst_width * channels;
	int k;

	unsigned int diff = 0;
	unsigned int temp = 0;

	// check every channel for differences. If just one of the channels has a difference above the threshold, then apply nearest neighbor. 

	for (k = 0; k < channels; k++) {
		temp = abs(dst[dst_index + k] - dst[dst_index + dst_stride * 3 + k]); // difference between two color channels
		if (temp > diff)
			diff = temp;
	}

	if (diff > threshold) { // nearest neighbor
		for (k = 0; k < channels; k++) {
			dst[dst_index + dst_stride + k] = dst[dst_index + k];
			dst[dst_index + dst_stride * 2 + k] = dst[dst_index + dst_stride * 3 + k];
		}
	}
	else { // linear
		int step;
		for (k = 0; k < channels; k++) {
			step = (dst[dst_index + k] - dst[dst_index + dst_stride * 3 + k]) / 3;
			dst[dst_index + dst_stride + k] = dst[dst_index + k] - step;
			dst[dst_index + dst_stride * 2 + k] = dst[dst_index + k] - (2 * step);
		}
	}
}


