#include "hip/hip_runtime.h"
// Author: Noah Van Der Weide
#include "hip/hip_runtime.h"
#include ""
#include "Upscale_CUDA.h"

#define THREADS_PER_BLOCK 64

__global__ void upscale_CUDA(unsigned char* dst, unsigned char* src, int src_width, int src_height, int src_channels, unsigned char threshold);

void upscale(unsigned char* src, unsigned char* dst, int src_height, int src_width, int dst_height, int dst_width, int channels, unsigned char threshold) {
    // initialize device variables
    unsigned char* dev_src, * dev_dst;

    // number of elements (if a picture has 3 channels, this is 3 * pixels)
    int dst_elements = dst_width * dst_height * channels;
    int src_elements = src_width * src_height * channels;

    // number of bytes each image will take
    int dst_size = dst_elements * sizeof(unsigned char);
    int src_size = src_elements * sizeof(unsigned char);

    // number of blocks to call in kernel. Max threads per block is usually 1024
    int blocks = (src_elements + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // allocate memory in GPU
    hipMalloc((void**)&dev_dst, dst_size);
    hipMalloc((void**)&dev_src, src_size);
    // used for shared memory if eventually implemented
    //hipMallocManaged(&dst, dst_size);
    //hipMallocManaged(&src, src_size);

    // copy data from CPU to GPU
    hipMemcpy(dev_dst, dst, dst_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_src, src, src_size, hipMemcpyHostToDevice);

    // start timer for performance evaluation
    //hipEventRecord(start);

    // call upscale function
    //upscale_CUDA<<<blocks, THREADS_PER_BLOCK>>>  (dev_dst, dev_src, src_elements, src_width, src_height, threshold); // <<<blocks, threads per block, shared mem>>>
    dim3 grid(src_width, src_height);
    Upscale_CUDA << <grid, 1 >> > (dev_dst, dev_src, src_width, src_height, channels, threshold);
    hipDeviceSynchronize();

    // end timer
    /*
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    */
    // copy data back from GPU to CPU
    hipMemcpy(dst, dev_dst, dst_size, hipMemcpyDeviceToHost);
    hipMemcpy(src, dev_src, dst_size, hipMemcpyDeviceToHost); // might not need this

    // free GPU
    hipFree(dev_dst);
    hipFree(dev_src);

}

__global__ void upscale_CUDA(unsigned char* dst, unsigned char* src, int src_width, int src_height, int src_channels, unsigned char threshold) {

    // not using shared memory right now
    // there is 48 KB of shared memory available.
    // images are typically more than that, so I'll have to think about how it could be implemented
    //extern __shared__ unsigned char pic[];

  //int pixel = blockIdx.x * blockdim.x + threadIdx.x;



    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // not relevant to code function, but shows how a thread could access a pixel in every channel.
    // pixel values are from 0 to 255.
    //for (int k = 0; k < channels; k++){
    //  img[idx + k];
    //}

    int dst_width = src_width * 3 - 2;
    //int dst_height = src_height * 3 - 2;

    //long int dst_elements = dst_width * dst_height * src_channels;
    //long int src_elements = src_width * src_height * src_channels;

    int src_stride = src_width * src_channels;
    int dst_stride = dst_width * src_channels;

    // if invalid location do nothing.
    //if (i >= dst_width || j >= dst_height) // is that width or width-1?
    if (i >= src_width || j >= src_height)
        return;

    // all channels for a pixel are grouped together. To access an adjacent pixel, you must add by the number of channels.
    for (int k = 0; k < src_channels; k++) {

        int dst_index = (j * 21 + i * 3) + k; // this is strictly for my predefined dst width and height (*3 -2)
        int src_index = (j * src_width + i) + k;

        // transfer known src values to dst
        // to access different channels, the number of elements of the src/dst image must be added to the respective array index.
        dst[dst_index] = src[src_index];

        // vertical comparison acts on src image and applies values to dst image
        int y_diff = src[src_index + src_stride] - src[src_index];
        if (y_diff < threshold) { // apply third-average
           // linear fill
            int step = y_diff / 3;
            dst[dst_index + dst_stride] = src[src_index] + step;
            dst[dst_index + 2 * dst_stride] = src[src_index] + step * 2;
        }
        else { // nearest neighbor
            dst[dst_index + dst_stride] = src[src_index];
            dst[dst_index + 2 * dst_stride] = src[src_index + src_stride];
        }

        __syncthreads();

        // horizontal
        // I know this is painfully inefficient. 
        int x_diff_0 = src[src_index] - src[src_index + src_channels];
        int x_diff_1 = dst[dst_index + dst_stride] - dst[dst_index + dst_stride + src_channels];
        int x_diff_2 = dst[dst_index + 2 * dst_stride] - dst[dst_index + 2 * dst_stride + src_channels];
        int step = 0;

        if (x_diff_0 < threshold) { // apply third-average
            // linear fill
            step = x_diff_0 / 3;
            dst[dst_index + 1] = src[src_index] + step;
            dst[dst_index + 2] = src[src_index] + step * 2;
        }
        else { // nearest neighbor
            dst[dst_index + src_channels] = src[src_index];
            dst[dst_index + 2 * src_channels] = src[src_index + src_channels];
        }

        if (x_diff_1 < threshold) { // apply third-average
            // linear fill
            step = x_diff_1 / 3;
            dst[dst_index + dst_stride + src_channels] = dst[dst_index + dst_stride] + step;
            dst[dst_index + dst_stride + 2 * src_channels] = dst[dst_index + dst_stride] + step * 2;
        }
        else { // nearest neighbor
            dst[dst_index + dst_stride + src_channels] = dst[dst_index + dst_stride];
            dst[dst_index + dst_stride + 2 * src_channels] = dst[dst_index + dst_stride + 3];
        }

        if (x_diff_2 < threshold) { // apply third-average
            // linear fill
            step = x_diff_2 / 3;
            dst[dst_index + 2 * dst_stride + src_channels] = dst[dst_index + 2 * dst_stride] + step;
            dst[dst_index + 2 * dst_stride + 2 * src_channels] = dst[dst_index + 2 * dst_stride] + step * 2;
        }
        else { // nearest neighbor
            dst[dst_index + 2 * dst_stride + src_channels] = dst[dst_index + 2 * dst_stride];
            dst[dst_index + 2 * dst_stride + 2 * src_channels] = dst[dst_index + 2 * dst_stride + 3];
        }
        __syncthreads();
    }
    __syncthreads();
}
